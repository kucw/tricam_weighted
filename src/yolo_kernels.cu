#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "hiprand.h"
#include "hipblas.h"
#include "lane_detection.cpp"
#include "unistd.h"
#include "utils.h"
extern "C" {
#include "network.h"
#include "detection_layer.h"
#include "cost_layer.h"
#include "utils.h"
#include "parser.h"
#include "box.h"
#include "convert_box.h"
#include "demorgan.h"
#include "image.h"
#include "thpool.h"
#include <sys/time.h>
#include <signal.h>
#include <fcntl.h>
#include <unistd.h>
#include <arpa/inet.h>
#include <sys/stat.h>
#include <math.h>
#include <stdio.h>
}

/* Change class number here */
#define CLS_NUM 20
#define RUN_TIMES 1

#ifdef OPENCV
#include "opencv2/highgui/highgui.hpp"
#include "opencv2/imgproc/imgproc.hpp"
extern "C" IplImage* image_to_Ipl(image img, int w, int h, int depth, int c, int step);
extern "C" image ipl_to_image(IplImage* src);
extern "C" void convert_yolo_detections(float *predictions, int classes, int num, int square, int side, int w, int h, float thresh, float **probs, box *boxes, int only_objectness);
extern "C" void draw_yolo(image im, int num, float thresh, box *boxes, float **probs);

extern "C" char *voc_names[];
extern "C" image voc_labels[];
extern "C" void draw_text(image a, char Text[], CvPoint TextPos);
#define RESULT_SIZE 55
static float **probs_left;
static float **probs_right;
static float **probs_up;
static box *boxes_left;
static box *boxes_right;
static box *boxes_up;
static network net;
static network net2;
static image in   ;
static image in_s ;
static image in_left;
static image in_right;
static image in_up;
static image det  ;
static image det_s;
static image det_left;
static image det_right;
static image det_up;
static image disp ;
static cv::VideoCapture cap;
static cv::VideoWriter cap_out;
static float demo_thresh = 0.2;
static float demo_thresh_low = 0.02;
static int w, h, depth, c, step= 0;
float FPS = 0;

int result[RESULT_SIZE] = {0};
int current = 0;
#define LEFT 1
#define MIDDLE 2
#define RIGHT 3
#define SINGLE 4
typedef struct ObjDetArg{
	image ROI;
	int draw;
}ODA;
int output;
int *control = (int*)malloc(sizeof(int));
int *traffic_mode = (int*)malloc(sizeof(int));

char mode[20];
char lane[20];
char fpss[20];
cv::Mat frames[1900];
struct timeval start_time;	

int right_falsepositive_map[450][450][5];
int left_falsepositive_map[450][450][5];

Rect rightbox_in_left[100];
Rect upbox_in_left[100];
Rect leftbox_in_left[100];
//int Rtree[10][300];	//assume there are 300 leaf box, M is 3, so level is 6



//========== control parameter ==================

//int frame_counter = 200; //demo frame counter
//int frame_counter = 1200;
int frame_counter = 0;
#define DRAW_CONVERT
#define DRAW_LOW_THRESHOLD_DETECTION
//#define DEMORGAN_RIGHT
//#define WEIGHTED_DEMORGAN_RIGHT
//#define WEIGHTED_POWER_DEMORGAN_RIGHT
//#define DEMORGAN_LEFT
//#define WEIGHTED_DEMORGAN_LEFT
#define WEIGHTED_POWER_DEMORGAN_LEFT
#define FRAME_BY_FRAME
//#define BOTH_DEMORGAN
//#define FALSE_POSITIVE_REMOVAL

//===============================================


//cv::Mat *frames = (cv::Mat*)malloc(sizeof(cv::Mat)*1800);

void *fetch_in_thread(void *Elastic)
{
	//int elastic = *((int*)Elastic);
	struct timeval now;
	
	gettimeofday(&now, NULL);
	int msec = (now.tv_sec - start_time.tv_sec)*1000 + (now.tv_usec - start_time.tv_usec)/1000;
	

	//frame_counter = msec/30;
	frame_counter++;
	IplImage frame = frames[frame_counter];

	if(step == 0)
	{
		w = frame.width;
		h = frame.height;
		c = frame.nChannels;
		depth= frame.depth; 
		step = frame.widthStep;
	}   

	
	int a, b, c;
	for(a=0; a<448; a++){
		for(b=0; b<448; b++){
			right_falsepositive_map[a][b][frame_counter%5] = 0;
			left_falsepositive_map[a][b][frame_counter%5] = 0;
		}
	}
	

	//output = lane_detection(&frame, result, RESULT_SIZE, current);

	in = ipl_to_image(&frame);
	rgbgr_image(in);
	in_s = resize_image(in, net.w, net.h);
	//in_left = crop_image(in,238,550,448,448);
	in_left = crop_image(in,238,400,448,448);	//reduce the car region
	in_right = crop_image(in,1130,630,448,448);
	in_up = crop_image(in,770,80,448,448);
	
	output = MIDDLE;
	if(output == MIDDLE){
		*control = 3;
		strcpy(lane, "");
		draw_box(in,238,550,676,998,0,0,0);
		draw_box(in,1130,630,1578,1078,0,0,0);
		draw_box(in,770,80,1218,528,0,0,0);
	}
	
	return 0;
}


int max_index(float *a, int n)
{
    if(n <= 0) return -1;
    int i, max_i = 0;
    float max = a[0];
    for(i = 1; i < n; ++i){
        if(a[i] > max){
            max = a[i];
            max_i = i;
        }
    }
    return max_i;
}


void *detect_in_thread_up(void *arg)
{
	ODA tmp = *((ODA*)arg);
	float nms = .4;
	detection_layer l = net.layers[net.n-1];
	//show_image(tmp.ROI,"123");
	float *X = tmp.ROI.data;
	float *predictions = network_predict(net, X);

	//free_image(tmp.ROI);
	convert_yolo_detections(predictions, l.classes, l.n, l.sqrt, l.side, 1, 1, demo_thresh_low, probs_up, boxes_up, 0);
	if (nms > 0) do_nms(boxes_up, probs_up, l.side*l.side*l.n, l.classes, nms);
#ifdef DRAW_LOW_THRESHOLD_DETECTION
	draw_detections(det, l.side*l.side*l.n, demo_thresh_low, boxes_up, probs_up, voc_names, voc_labels, CLS_NUM,tmp.draw);
#else
	draw_detections(det, l.side*l.side*l.n, demo_thresh, boxes_up, probs_up, voc_names, voc_labels, CLS_NUM,tmp.draw);
#endif

#ifdef DRAW_CONVERT
	int i;
	int num = l.side*l.side*l.n; 
	for(i = 0; i < num; ++i){
		int obj_class = max_index(probs_up[i], CLS_NUM);
		if(voc_names[obj_class] == "car"){
			float prob_up = probs_up[i][obj_class];
			if(prob_up > demo_thresh_low){
				convert_upbox_to_leftROI(det, prob_up, boxes_up, voc_labels, i, obj_class, CLS_NUM);
				convert_upbox_to_rightROI(det, prob_up, boxes_up, voc_labels, i, obj_class, CLS_NUM);
			}
		}
	}
#endif

	//print MODE
	printf("\033[2J");
	printf("\033[1;1H");
	printf("\nframe_counter: %d\n", frame_counter);

	//print FPS
	printf("\nFPS:%.0f\n",FPS);
	sprintf(fpss, "FPS: %.0f", FPS);
	printf("Object:\n\n");
	

	return 0;
}

void *detect_in_thread_left(void *arg)
{
	ODA tmp = *((ODA*)arg);
	float nms = .4;
	detection_layer l = net.layers[net.n-1];
	//show_image(tmp.ROI,"123");
	float *X = tmp.ROI.data;
	float *predictions = network_predict(net, X);

	//free_image(tmp.ROI);
	convert_yolo_detections(predictions, l.classes, l.n, l.sqrt, l.side, 1, 1, demo_thresh_low, probs_left, boxes_left, 0);
	if (nms > 0) do_nms(boxes_left, probs_left, l.side*l.side*l.n, l.classes, nms);

#ifdef FALSE_POSITIVE_REMOVAL
	//Remove false positive
	int i;
	int num = l.side*l.side*l.n; 
	for(i = 0; i < num; ++i){
		int obj_class_left = max_index(probs_left[i], CLS_NUM);
		if(voc_names[obj_class_left] == "car"){
			float prob_left = probs_left[i][obj_class_left];
			if(prob_left > demo_thresh){
				int box_left[5];
				int x, y, j, count = 0, box_left_area = 0;
				get_leftbox_in_leftROI(det, box_left, prob_left, boxes_left, i);
				box_left_area = (box_left[1] - box_left[0]) * (box_left[3] - box_left[2]);	
				
				for(y=box_left[2]; y<box_left[3]; y++){
					for(x=box_left[0]; x<box_left[1]; x++){
						int buffer_count = 0;
						for(j=0; j<5; j++){
							if(left_falsepositive_map[y-550][x-238][j] == 1)
								buffer_count++;
						}
						if(buffer_count >= 3)
							count++;
						left_falsepositive_map[y-550][x-238][frame_counter%5] = 1;
					}
				}
				
				float area = (float)count/(float)box_left_area;
				
				if(area > 0.6){	
					float rgb[3];
					get_normal_box_color(rgb, obj_class_left, CLS_NUM);
					if(voc_labels) draw_label(det, box_left[2] + box_left[4], box_left[0], voc_labels[obj_class_left], rgb);
					draw_box_width(det, box_left[0], box_left[2], box_left[1], box_left[3], box_left[4], rgb[0], rgb[1], rgb[2]);
					char Text[30];
					sprintf(Text, "%.2f", prob_left);
					IplImage *text = image_to_Ipl(det,det.w,det.h,IPL_DEPTH_8U,det.c,det.w*det.c);			
					CvFont font2;
					CvPoint TextPos;
					TextPos.x = (box_left[0]+box_left[1])/2-50; TextPos.y = box_left[2];
					cvInitFont(&font2 , CV_FONT_HERSHEY_SIMPLEX , 1 , 1 , 1 , 3 , CV_AA);
					cvPutText(text , Text , TextPos , &font2 , CV_RGB(0, 133, 255));	
					image d = ipl_to_image(text);  
					memcpy(det.data,d.data,det.h*det.w*det.c*sizeof(float));
					free_image(d);
					cvReleaseImage(&text);										
				}
			}
		}
	}
#elif defined(DRAW_LOW_THRESHOLD_DETECTION)
	draw_detections(det, l.side*l.side*l.n, demo_thresh_low, boxes_left, probs_left, voc_names, voc_labels, CLS_NUM, tmp.draw);
#else
	draw_detections(det, l.side*l.side*l.n, demo_thresh, boxes_left, probs_left, voc_names, voc_labels, CLS_NUM, tmp.draw);
#endif



#ifdef DRAW_CONVERT
	int i;
	int num = l.side*l.side*l.n; 
	for(i = 0; i < num; ++i){
		int obj_class = max_index(probs_left[i], CLS_NUM);
		if(voc_names[obj_class] == "car"){
			float prob_left = probs_left[i][obj_class];
			if(prob_left > demo_thresh_low){
				convert_leftbox_to_rightROI(det, prob_left, boxes_left, voc_labels, i, obj_class, CLS_NUM);
				convert_leftbox_to_upROI(det, prob_left, boxes_left, voc_labels, i, obj_class, CLS_NUM);
			}
		}
	}
#endif
	
	//print MODE
	printf("\033[2J");
	printf("\033[1;1H");
	printf("\nframe_counter: %d\n", frame_counter);

	//print FPS
	printf("\nFPS:%.0f\n",FPS);
	sprintf(fpss, "FPS: %.0f", FPS);
	printf("Object:\n\n");
	

	return 0;
}


void *detect_in_thread_right(void *arg)
{
	ODA tmp = *((ODA*)arg);
	float nms = .4;
	detection_layer l = net.layers[net.n-1];
	//show_image(tmp.ROI,"123");
	float *X = tmp.ROI.data;
	float *predictions = network_predict(net, X);

	//free_image(tmp.ROI);
	convert_yolo_detections(predictions, l.classes, l.n, l.sqrt, l.side, 1, 1, demo_thresh_low, probs_right, boxes_right, 0);
	if (nms > 0) do_nms(boxes_right, probs_right, l.side*l.side*l.n, l.classes, nms);
	
	convert_allrightbox_to_leftROI(det, rightbox_in_left, probs_right, boxes_right, l.side*l.side*l.n, CLS_NUM);


#ifdef FALSE_POSITIVE_REMOVAL	
	//Remove false positive
	int i;
	int num = l.side*l.side*l.n; 
	for(i = 0; i < num; ++i){
		int obj_class_right = max_index(probs_right[i], CLS_NUM);
		if(voc_names[obj_class_right] == "car"){
			float prob_right = probs_right[i][obj_class_right];
			if(prob_right > demo_thresh){
				int box_right[5];
				int x, y, j, count = 0, box_right_area = 0;
				get_rightbox_in_rightROI(det, box_right, prob_right, boxes_right, i);
				box_right_area = (box_right[1] - box_right[0]) * (box_right[3] - box_right[2]);	
				
				for(y=box_right[2]; y<box_right[3]; y++){
					for(x=box_right[0]; x<box_right[1]; x++){
						int buffer_count = 0;
						for(j=0; j<5; j++){
							if(right_falsepositive_map[y-630][x-1130][j] == 1)
								buffer_count++;
						}
						if(buffer_count >= 3)
							count++;
						right_falsepositive_map[y-630][x-1130][frame_counter%5] = 1;
					}
				}
				
				float area = (float)count/(float)box_right_area;
				
				if(area > 0.6){	
					float rgb[3];
					get_normal_box_color(rgb, obj_class_right, CLS_NUM);
					if(voc_labels) draw_label(det, box_right[2] + box_right[4], box_right[0], voc_labels[obj_class_right], rgb);
					draw_box_width(det, box_right[0], box_right[2], box_right[1], box_right[3], box_right[4], rgb[0], rgb[1], rgb[2]);
					char Text[30];
					sprintf(Text, "%.2f", prob_right);
					IplImage *text = image_to_Ipl(det,det.w,det.h,IPL_DEPTH_8U,det.c,det.w*det.c);			
					CvFont font2;
					CvPoint TextPos;
					TextPos.x = (box_right[0]+box_right[1])/2-50; TextPos.y = box_right[2];
					cvInitFont(&font2 , CV_FONT_HERSHEY_SIMPLEX , 1 , 1 , 1 , 3 , CV_AA);
					cvPutText(text , Text , TextPos , &font2 , CV_RGB(0, 133, 255));	
					image d = ipl_to_image(text);  
					memcpy(det.data,d.data,det.h*det.w*det.c*sizeof(float));
					free_image(d);
					cvReleaseImage(&text);										
				}
			}
		}
	}
#elif defined(DRAW_LOW_THRESHOLD_DETECTION)
	draw_detections(det, l.side*l.side*l.n, demo_thresh_low, boxes_right, probs_right, voc_names, voc_labels, CLS_NUM, tmp.draw);
#else
	draw_detections(det, l.side*l.side*l.n, demo_thresh, boxes_right, probs_right, voc_names, voc_labels, CLS_NUM, tmp.draw);
#endif


#ifdef DRAW_CONVERT
	int i;
	int num = l.side*l.side*l.n; 
	for(i = 0; i < num; ++i){
		int obj_class = max_index(probs_right[i], CLS_NUM);
		if(voc_names[obj_class] == "car"){
			float prob_right = probs_right[i][obj_class];
			if(prob_right > demo_thresh_low){
				convert_rightbox_to_leftROI(det, prob_right, boxes_right, voc_labels, i, obj_class, CLS_NUM);
				convert_rightbox_to_upROI(det, prob_right, boxes_right, voc_labels, i, obj_class, CLS_NUM);
			}
		}
	}
#endif

	//print MODE
	printf("\033[2J");
	printf("\033[1;1H");
	printf("\nframe_counter: %d\n", frame_counter);

	//print FPS
	printf("\nFPS:%.0f\n",FPS);
	sprintf(fpss, "FPS: %.0f", FPS);
	printf("Object:\n\n");
	

	return 0
		;
}

extern "C" void demo_yolo(char *cfgfile, char *weightfile, float thresh, int cam_index, char *videofile, char *version)
{
	//demo_thresh = thresh;
	printf("YOLO demo\n");
	net = parse_network_cfg(cfgfile);
	if(weightfile){
		load_weights(&net, weightfile);
	}
	set_batch_network(&net, 1);

	int total_frame;

	srand(2222222);
	if(cam_index != -1)
	{
		cv::VideoCapture cam(cam_index);
		cap = cam;
		//if(!cap.isOpened()) error("Couldn't connect to webcam.\n");
	}
	else 
	{
		printf("Video File name is: %s\n", videofile);
		cv::VideoCapture videoCap(videofile);
		cap = videoCap;
		//if(!cap.isOpened()) error("Couldn't read video file.\n");

		cv::Size S = cv::Size((int)videoCap.get(CV_CAP_PROP_FRAME_WIDTH), (int)videoCap.get(CV_CAP_PROP_FRAME_HEIGHT));
		//Preload all frames

		total_frame= (int)videoCap.get(CV_CAP_PROP_FRAME_COUNT);

		fprintf(stderr, "preLoad...\n");
		int frame_number = 0;
		for(;frame_number < total_frame;){
			cap >> frames[frame_number++];
			//if(cv::waitKey(0)>=0)break;
		}
		printf("Load OK.\n");
		
			
		//cv::VideoWriter outputVideo("out.avi", CV_FOURCC('D','I','V','X'), videoCap.get(CV_CAP_PROP_FPS), S, true);
		//if(!outputVideo.isOpened()) error("Couldn't write video file.\n");
		//cap_out = outputVideo;
	}
	
	int i, j, k;
	for(i=0; i<448; i++){
		for(j=0; j<448; j++){
			for(k=0; k<5; k++){
				right_falsepositive_map[i][j][k] = 0;
				left_falsepositive_map[i][j][k] = 0;
			}
		}
	}

	detection_layer l = net.layers[net.n-1];
	gettimeofday(&start_time, NULL);

	boxes_left = (box *)calloc(l.side*l.side*l.n, sizeof(box));
	boxes_right = (box *)calloc(l.side*l.side*l.n, sizeof(box));
	boxes_up = (box *)calloc(l.side*l.side*l.n, sizeof(box));
	probs_left = (float **)calloc(l.side*l.side*l.n, sizeof(float *));
	probs_right = (float **)calloc(l.side*l.side*l.n, sizeof(float *));
	probs_up = (float **)calloc(l.side*l.side*l.n, sizeof(float *));
	for(j = 0; j < l.side*l.side*l.n; ++j){
		probs_left[j] = (float *)calloc(l.classes, sizeof(float *));
		probs_right[j] = (float *)calloc(l.classes, sizeof(float *));
		probs_up[j] = (float *)calloc(l.classes, sizeof(float *));
	}
	threadpool thpool_cpu = thpool_init(4);
	threadpool thpool_gpu = thpool_init(1);
	//pthread_t fetch_thread;
	//pthread_t detect_thread;
	ODA *arg = (ODA*)malloc(sizeof(ODA));
	fetch_in_thread(0);
	det = in;
	det_s = in_s;
	det_left = in_left;
	det_right = in_right;
	det_up = in_up;
	fetch_in_thread(arg);
	detect_in_thread_left(arg);
	disp = det;
	det = in;
	det_s = in_s;
	det_left = in_left;
	det_right = in_right;
	det_up = in_up;
	for (int k = 0; k < RUN_TIMES; ++k){
		//frame_counter = 250;
		do {
			struct timeval tval_before, tval_after, tval_result;	
			gettimeofday(&tval_before, NULL);
	
			if(*control == 3){
				thpool_add_work(thpool_cpu,fetch_in_thread,0);
				arg->ROI = det_left;
				arg->draw = 1;
				thpool_add_work(thpool_gpu,detect_in_thread_left,arg);
				thpool_wait(thpool_gpu);
				arg->ROI = det_right;
				arg->draw = 2;
				thpool_add_work(thpool_gpu,detect_in_thread_right,arg);
				thpool_wait(thpool_gpu);
				arg->ROI = det_up;
				arg->draw = 3;
				thpool_add_work(thpool_gpu,detect_in_thread_up,arg);
				thpool_wait(thpool_gpu);
				thpool_wait(thpool_cpu);

				

#ifdef WEIGHTED_DEMORGAN_LEFT
				Weighted_Demorgan_left(det, demo_thresh, demo_thresh_low, probs_right, probs_left, probs_up, boxes_right, boxes_left, boxes_up, voc_names, voc_labels, CLS_NUM, l.side*l.side*l.n);
#endif
#ifdef WEIGHTED_POWER_DEMORGAN_LEFT
				Weighted_Demorgan_Power_left(det, demo_thresh, demo_thresh_low, probs_right, probs_left, probs_up, boxes_right, boxes_left, boxes_up, voc_names, voc_labels, CLS_NUM, l.side*l.side*l.n, frame_counter, left_falsepositive_map);
#endif
#ifdef DEMORGAN_LEFT
				Demorgan_left(det, demo_thresh, demo_thresh_low, probs_right, probs_left, probs_up, boxes_right, boxes_left, boxes_up, voc_names, voc_labels, CLS_NUM, l.side*l.side*l.n);
#endif


#ifdef WEIGHTED_DEMORGAN_RIGHT
				Weighted_Demorgan_right(det, demo_thresh, demo_thresh_low, probs_right, probs_left, probs_up, boxes_right, boxes_left, boxes_up, voc_names, voc_labels, CLS_NUM, l.side*l.side*l.n);
#endif
#ifdef WEIGHTED_POWER_DEMORGAN_RIGHT
				Weighted_Demorgan_Power_right(det, demo_thresh, demo_thresh_low, probs_right, probs_left, probs_up, boxes_right, boxes_left, boxes_up, voc_names, voc_labels, CLS_NUM, l.side*l.side*l.n, frame_counter, right_falsepositive_map);
#endif
#ifdef DEMORGAN_RIGHT
				Demorgan_right(det, demo_thresh, demo_thresh_low, probs_right, probs_left, probs_up, boxes_right, boxes_left, boxes_up, voc_names, voc_labels, CLS_NUM, l.side*l.side*l.n);
#endif

#ifdef BOTH_DEMORGAN
				Weighted_Demorgan_Power_right(det, demo_thresh, demo_thresh_low, probs_right, probs_left, probs_up, boxes_right, boxes_left, boxes_up, voc_names, voc_labels, CLS_NUM, l.side*l.side*l.n);
				Weighted_Demorgan_Power_left(det, demo_thresh, demo_thresh_low, probs_right, probs_left, probs_up, boxes_right, boxes_left, boxes_up, voc_names, voc_labels, CLS_NUM, l.side*l.side*l.n);
				Weighted_Demorgan_Power_up(det, demo_thresh, demo_thresh_low, probs_right, probs_left, probs_up, boxes_right, boxes_left, boxes_up, voc_names, voc_labels, CLS_NUM, l.side*l.side*l.n);
				Demorgan_right(det, demo_thresh, demo_thresh_low, probs_right, probs_left, probs_up, boxes_right, boxes_left, boxes_up, voc_names, voc_labels, CLS_NUM, l.side*l.side*l.n);
				Demorgan_left(det, demo_thresh, demo_thresh_low, probs_right, probs_left, probs_up, boxes_right, boxes_left, boxes_up, voc_names, voc_labels, CLS_NUM, l.side*l.side*l.n);
				Demorgan_up(det, demo_thresh, demo_thresh_low, probs_right, probs_left, probs_up, boxes_right, boxes_left, boxes_up, voc_names, voc_labels, CLS_NUM, l.side*l.side*l.n);
#endif
				
			}

			
    		//save_image(disp, "test");
			show_image_and_text(disp, "YOLO", mode, lane, fpss);
			free_image(disp);
#ifdef FRAME_BY_FRAME
			cvWaitKey(0);
#else
			cvWaitKey(1);
#endif
			thpool_wait(thpool_cpu);
			thpool_wait(thpool_gpu);
			disp  = det;
			free_image(det_s);
			free_image(det_left);
			free_image(det_right);
			free_image(det_up);
			det   = in;
			det_s = in_s;
			det_left = in_left;
			det_right = in_right;
			det_up = in_up;
			gettimeofday(&tval_after, NULL);
			timersub(&tval_after, &tval_before, &tval_result);
			float curr = 1000000.f/((long int)tval_result.tv_usec);
			FPS = .9*FPS + .1*curr;

		}while(frame_counter < total_frame - 30);
		struct timeval stop_time;
		gettimeofday(&stop_time, NULL);
		double msec = (stop_time.tv_sec - start_time.tv_sec)*1000 + (stop_time.tv_usec - start_time.tv_usec)/1000;
		printf("1 frame = %f ms\n", (double)msec/(double)frame_counter);

	}
}
#else
extern "C" void demo_yolo(char *cfgfile, char *weightfile, float thresh, int cam_index){
	fprintf(stderr, "YOLO demo needs OpenCV for webcam images.\n");
}
#endif

